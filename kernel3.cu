
#include <hip/hip_runtime.h>
__global__ void kernel3(double *tab, double *colk, int k, int r, int n) 
{

int i = blockDim.x * blockIdx.x + threadIdx.x;
int j = blockIdx.y * blockIdx.y + threadIdx.y;
__shared__ double w[16];
if (threadIdx.y == 0 && threadIdx.x < 16)
{
w[threadIdx.x] = colk[blockIdx.y * blockDim.y + threadIdx.x];
}
__syncthreads();
printf("%f ", tab[i * (n+1) + j]);
if (j == r) return;
//tab[i * (n+1) + j] = tab[i * (n+1) + j] - w[threadIdx.y] * tab[r * (n+1) + i];
tab[i * (n+1) + j] = tab[i * (n+1) + j] - tab[i * (n+1) + k] * tab[r * (n+1) + j];

}