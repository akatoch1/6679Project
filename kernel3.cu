
#include <hip/hip_runtime.h>
__global__ void kernel3(double *tab, double *colk, int k, int r, int n, int m) 
{
int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    __shared__ double w[16];
    if (j == r) {
        return;
    }
    if (i > m+1) {
        return;
    }  
    if (j > m+1) {
        return;
    }    
    if (threadIdx.y == 0 && threadIdx.x < 16)
    {
        w[threadIdx.x] = colk[blockIdx.y * blockDim.y + threadIdx.x];
    }
    __syncthreads();
    
    if (j == r) {
        return;
    }
    
    tab[j * (n+1) + i] = tab[j * (n+1) + i] - w[threadIdx.y] * tab[(r * (n+1) + i)];
}