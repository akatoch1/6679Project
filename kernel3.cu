
#include <hip/hip_runtime.h>
__global__ void kernel3(double *tab, double *colk, int k, int r, int n) 
{
int i = blockDim.x * blockIdx.x + threadIdx.x;
int j = blockIdx.y * blockIdx.y + threadIdx.y;
__shared__ double w[16];
if (threadIdx.y == 0 && threadIdx.x < 16)
{
w[threadIdx.x] = colk[blockIdx.y * blockDim.y + threadIdx.x];
}
__syncthreads();
if (j == r) return;
tab[j * (n+1) + i] = tab[j * (n+1) + i] - w[threadIdx.y] * tab[r * (n+1) + i];
}