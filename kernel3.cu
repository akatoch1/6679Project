
#include <hip/hip_runtime.h>
__global__ void kernel3(double **tab, double *colk, int k, int r) 
{
int i = blockDim.x * blockIdx.x + threadIdx.x;
int j = blockIdx.y * blockIdx.y + threadIdx.y;
__shared__ double w[16];
if (threadIdx.y == 0 && threadIdx.x < 16)
{
w[threadIdx.x] = colk[blockIdx.y * blockDim.y + threadIdx.x];
}
__syncthreads();
if (j == r) return;
tab[j][i] = tab[j][i] - w[threadIdx.y] * tab[r][i];
}