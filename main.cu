#include <stdio.h>
#include "kernel1.cu"
#include "kernel2.cu"
#include "kernel3.cu"
#include "kernel4.cu"
#include "support.h"
void printMatrix(double *m, int r, int c) {
  for (int i = 0; i < r; i++)
    {
      for (int j = 0; j < c; j++)
      {
        printf("%f ", m[i*c + j]);
	}	    
      printf("\n");
    }
}
void printMatrix2(double **m, int r, int c) {
  for (int i = 0; i < r; i++)
    {
      for (int j = 0; j < c; j++)
      {
        printf("%f ", m[i][j]);
        }
      printf("\n");
    }
}

int main(int argc, char**argv) {


    unsigned int m; // number of constraints
    unsigned int n; // number of variables (no slack vars)
     // b = highest value of constraint value 
    unsigned int b;
    //unsigned int hb;

    //m = atoi(argv[1]);
    //n = atoi(argv[2]);
    //b = atoi(argv[3]);
    m = 2;
    n = 2;
    
    hipError_t cuda_ret;
    // Initialize host variables
    double* xB_h = (double*) malloc( sizeof(double)*m ); // index's of the variables in the Basis.
    for (unsigned int i=0; i < m; i++) { xB_h[i] = n+i; }
    
    double* xN_h = (double*) malloc( sizeof(double)*n ); // index's of the variables not in the Basis.
    for (unsigned int i=0; i < n; i++) { xN_h[i] = i; }
    
    double* cB_h = (double*) malloc( sizeof(double)*m ); // C values for the basic variables
    for (unsigned int i=0; i < m; i++) { cB_h[i] = 0; }
    
    
    double* cN_h = (double*) malloc( sizeof(double)*n ); // C values for the non-basic variables
    for (unsigned int i=0; i < n; i++) { cN_h[i] = (rand()%10-5); }
    cN_h[0] = 40;
    cN_h[1] = 30;
    double* b_h = (double*) malloc( sizeof(double)*m ); // Right hand side values 
    for (unsigned int i=0; i < m; i++) { b_h[i] = rand()%b; }
    b_h[0] = 12;
    b_h[1] = 16;
    double* svec = (double*) malloc( sizeof(double)*(m+1));
    



    //double* B_h = (double*) malloc( sizeof(double)*m*m ); // Constraint coefficents of basic variables
    double* B_h[m];
      for (int i = 0; i < m; i++) {
        B_h[i] = (double*)malloc(m * sizeof(double));
      }  
    for (unsigned int i=0; i < m; i++) { 
        for (unsigned int j=0; j < m; j++) { 
            if (i==j) {
                B_h[i][j] = 1;
            } else {
                B_h[i][j] = 0;
            }
        }
    }


    double* cBB	= (double*) malloc( sizeof(double)*m);
    for	(int i = 0; i < m; i++) {
    	double z = 0;
    	for (int j = 0; j < m; j++) {
            z += cB_h[j] * B_h[j][i];
     	}
	cBB[i] = z;
    }
    double z = 0;
    for (int i = 0; i < m; i++) {
    	z += cBB[i] * b_h[i];
    }
    svec[0] = z;
    for (int i = 1; i < m+1; i++) {
    	double sum = 0;
	for (int j = 0; j < m; j++) {
	    sum += B_h[i-1][j] * b_h[j];
	}
	svec[i] = sum;
    }   


       
    //double* N_h = (double*) malloc( sizeof(double)*m*n ); // Constraint coefficents of non-basic variables
    double* N_h[m];
    for (int i = 0; i < m; i++) {
        N_h[i] = (double*)malloc(n * sizeof(double));
    }
   
    fflush(stdout);
    for (unsigned int i=0; i < m; i++) { 
        for (unsigned int j=0; j < n; j++) { 
            N_h[i][j] = (rand()%10-5);
        }
    }
    N_h[0][0] = 1;
    N_h[0][1] = 1;
    N_h[1][0] = 2;
    N_h[1][1] = 1;
    
    
    double* tab_h = (double *) malloc(((n+1) * (m+1)) * sizeof(double));
    double* columnk_h = (double *) malloc((m+1) * sizeof(double));
 
 
    // Assign vals for 0,0
    // Assign vals for m,0
    // Assign vals for 0,n
    // Assign vals for m,n
    double* smat[m+1];
    for (int i = 0; i < m+1; i++) {
        smat[i] = (double*) malloc(n * sizeof(double));
    }
    //update first row of smat
    for (int i = 0; i < n; i++) {
        double val = 0;
        for (int j = 0; j < m; j++) {
            val += cBB[j] * N_h[j][i];
        }
        smat[0][i] = val - cN_h[i];
    }
    for (int i = 1; i < m+1; i++) {

        for (int j = 0; j < n; j++) {
	    double val = 0;
	    for (int k = 0; k < m; k++) {
	    	
		val += B_h[i-1][k] * N_h[k][j];
		
	    	
	    }
	    smat[i][j] = val;
        }
    }

    //update tab
  //  for (int i = 0; i < m+1; i++) {
   // 	tab_h[i * (n+1)] = svec[i];
  //  }    
    //printMatrix2(smat, m+1, n);
    for (int i = 0; i < m+1; i++) {
    	for (int j = 0; j < n+1; j++) {
	    tab_h[i * (n+1) + (j+1)] = smat[i][j];
	}   
    }	
    for (int i = 0; i < m+1; i++) {
        tab_h[i * (n+1)] = svec[i];
    }
    double* objLine_h = (double*) malloc( sizeof(double)*(n+1)); // Objective line used to determine k
    
    int k_h;
    


    double* theta_h = (double*) malloc( sizeof(double)*(m+1)); // Ratio of right-hand side to k row

    // Allocate device variables
    double* tab_d;
    cuda_ret = hipMalloc((void**) &tab_d, sizeof(double)*(m+1)*(n+1));
	if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
    

 
    double* xB_d;
    cuda_ret = hipMalloc((void**) &xB_d, sizeof(double)*m);
	if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
    

	
    int k_d;
    //cuda_ret = hipMalloc((void**) &k_d, sizeof(int));
//	if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    int r_d;
    cuda_ret = hipMalloc((void**) &r_d, sizeof(int));
	if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    double* columnk_d;
    cuda_ret = hipMalloc((void**) &columnk_d, sizeof(double)*(m+1));
	if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    double* theta_d;
    cuda_ret = hipMalloc((void**) &theta_d, sizeof(double)*(m+1));
	if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    
    // Copy tableau to Device

    cuda_ret = hipMemcpy(theta_d, theta_h, sizeof(double)*(m+1), hipMemcpyHostToDevice);
        if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device");

    cuda_ret = hipMemcpy(tab_d, tab_h, sizeof(double)*(m+1)*(n+1), hipMemcpyHostToDevice);
	if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device");

    cuda_ret = hipMemcpy(xB_d, xB_h, sizeof(double)*m, hipMemcpyHostToDevice);

	if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device");
   
    //printMatrix(tab_h, m+1, n+1);	
    // Iterate until optimal or infeasible/unbounded is found
    bool continueVar = true;
    const int THREADS_PER_BLOCK = 16;    
    while (continueVar == true) {
        // Copy first line of the Simplex tableau to host
        for (unsigned int i=0; i < n+1; i++) {objLine_h[i]= tab_h[i];}


        
        hipDeviceSynchronize();

        // Find index of entering variable k
        double minValue = 1.0;
        int minIndex = 0;
        for (unsigned int i=1; i < n+1; i++) {
            if (objLine_h[i] < minValue) {
                minValue = objLine_h[i];
                minIndex = i;
            }
        }
        if (minValue>=0) {
            printf("Optimal Value");
            continueVar = false;
            break;
        } else {
            k_h = minIndex;
        }





        hipDeviceSynchronize();
	printMatrix(tab_h, m+1, n+1);
	printf("\n");
        // Kernel 1 to process ratio column

        const unsigned int numBlocks1 = m/THREADS_PER_BLOCK + 1;
        dim3 gridDim(numBlocks1, 1, 1), blockDim(THREADS_PER_BLOCK, 1, 1);
        kernel1<<<gridDim, blockDim>>>(tab_d, theta_d, columnk_d, k_h, n);

        hipDeviceSynchronize();
	
        // Copy ratio column to host
        cuda_ret = hipMemcpy(theta_h, theta_d, sizeof(double)*(m+1), hipMemcpyDeviceToHost);
    	if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to host");
	
	cuda_ret = hipMemcpy(columnk_h, columnk_d, sizeof(double)*(m+1), hipMemcpyDeviceToHost);
        if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to host");        

	cuda_ret = hipMemcpy(columnk_d, columnk_h, sizeof(double)*(m+1), hipMemcpyHostToDevice);
        if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to host");
	// Find the index of the leaving variable r
        minValue = 10000.0;
        minIndex = 0;
	int r_h;
        for (unsigned int i=1; i < m+1; i++) {
            if (theta_h[i] < minValue && theta_h[i] > 0) {
                minValue = theta_h[i];
                minIndex = i;
            }
        }
        if (minValue==10000.0) {
            printf("unbounded");
            continueVar = false;
            break;
        } else {
            r_h = minIndex;
        }
        // Copy index r to device

	//printf("%d\n", r_h);
        // Kernel 2 to update the line r of the Simplex tableau
        const unsigned int numBlocks2 = n/THREADS_PER_BLOCK + 1;
        dim3 gridDim2(numBlocks2, 1, 1);
	dim3 blockDim2(THREADS_PER_BLOCK, 1, 1);

        kernel2<<<gridDim2, blockDim2>>>(tab_d, columnk_d, k_h, r_h, n);
	
        hipDeviceSynchronize();
	
	cuda_ret = hipMemcpy(tab_h, tab_d, sizeof(double)*(m+1)*(n+1), hipMemcpyDeviceToHost);
        if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device");
	
	printMatrix(tab_h, m+1,	 n+1);
	printf("\n");
	cuda_ret = hipMemcpy(tab_d, tab_h, sizeof(double)*(m+1)*(n+1), hipMemcpyHostToDevice);
        if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device");
	
	cuda_ret = hipMemcpy(columnk_h, columnk_d, sizeof(double)*(m+1), hipMemcpyDeviceToHost);
	if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device");	
	
	cuda_ret = hipMemcpy(columnk_d, columnk_h, sizeof(double)*(m+1), hipMemcpyHostToDevice);
	if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device");	
	
        // Kernel 3 to update Simplex tableau
        const unsigned int numBlocksX3 = m/THREADS_PER_BLOCK + 1;
        const unsigned int numBlocksY3 = n/THREADS_PER_BLOCK + 1;
        dim3 gridDim3(numBlocksX3, numBlocksY3), blockDim3(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
	
        kernel3<<<gridDim3, blockDim3>>>(tab_d, columnk_d, k_h, r_h, n, m);
	printf("\n");
        hipDeviceSynchronize();
	cuda_ret = hipMemcpy(tab_h, tab_d, sizeof(double)*(m+1)*(n+1), hipMemcpyDeviceToHost);
        if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device");



	printMatrix(tab_h, m+1,n+1);
	printf("\n");
        // Kernel 4 to Update column k of the Simplex Tableau
	cuda_ret = hipMemcpy(tab_d, tab_h, sizeof(double)*(m+1)*(n+1), hipMemcpyHostToDevice);
        if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device");	

	cuda_ret = hipMemcpy(columnk_h, columnk_d, sizeof(double)*(m+1), hipMemcpyDeviceToHost);
        if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device");

//	printMatrix(columnk_h, 1, m+1);
//	printf("%d\n", k_h);
//	printf("%d\n", r_h);

        cuda_ret = hipMemcpy(columnk_d, columnk_h, sizeof(double)*(m+1), hipMemcpyHostToDevice);
        if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device");
	
        const unsigned int numBlocks4 = n/THREADS_PER_BLOCK + 1;
        dim3 gridDim4(numBlocks4, 1), blockDim4(THREADS_PER_BLOCK, 1);
        kernel4<<<gridDim4, blockDim4>>>(tab_d, columnk_d, k_h, r_h, n);
		
        hipDeviceSynchronize();
    	cuda_ret = hipMemcpy(tab_h, tab_d, sizeof(double)*(m+1)*(n+1), hipMemcpyDeviceToHost);
        if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device");
       	printMatrix(tab_h, m+1, n+1);
	printf("############################################\n");
    }
    // Calculate optimal value and return it
    //cuda_ret = hipMemcpy(objLine_h, objLine_d, sizeof(double)*n, hipMemcpyDeviceToHost);
    //if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to host");
    
}
