#include <stdio.h>
#include "kernel1.cu"
#include "kernel2.cu"
#include "kernel3.cu"
#include "kernel4.cu"

int main(int argc, char**argv) {
    unsigned int m; // number of constraints
    unsigned int n; // number of variables (no slack vars)
    unsigned int b; // b = highest value of constraint value 
   
    m = atoi(argv[0]);
    n = atoi(argv[1]);
    b = atoi(argv[2]);
    
    // Initialize host variables
    double* xB_h = (double*) malloc( sizeof(double)*m );
    for (unsigned int i=0; i < m; i++) { xB_h[i] = n+1+i; }

    double* xN_h = (double*) malloc( sizeof(double)*n );
    for (unsigned int i=0; i < n; i++) { xB_h[i] = i+1; }

    double* cB_h = (double*) malloc( sizeof(double)*m );
    for (unsigned int i=0; i < m; i++) { cB_h[i] = 0; }

    double* cN_h = (double*) malloc( sizeof(double)*n );
    for (unsigned int i=0; i < n; i++) { cN_h[i] = (rand()%10-5); }

    double* b_h = (double*) malloc( sizeof(double)*m );
    for (unsigned int i=0; i < m; i++) { b_h[i] = rand()%b; }

    double* B_h = (double*) malloc( sizeof(double)*m*m );
    for (unsigned int i=0; i < m; i++) { 
        for (unsigned int j=0; j < m; j++) { 
            if (i==j) {
                B_h[i][j] = 1;
            } else {
                B_h[i][j] = 0;
            }
        }
    }

    double* N_h = (double*) malloc( sizeof(double)*m*n );
    for (unsigned int i=0; i < m; i++) { 
        for (unsigned int j=0; j < n; j++) { 
            B_h[i][j] = (rand()%10-5);
        }
    }

    double* tab_h = (double*) malloc( sizeof(double)*(m+1)*(n+1));
    // Assign vals for 0,0
    // Assign vals for m,0
    // Assign vals for 0,n
    // Assign vals for m,n

    double* objLine_h = (double*) malloc( sizeof(double)*n);

    int* k_h = malloc(sizeof(int));
    
    int* r_h = malloc(sizeof(int));

    double* theta_h = (double*) malloc( sizeof(double)*(m+1));

    // Allocate device variables
    double* tab_d;
    cuda_ret = hipMalloc((void**) &tab_d, sizeof(double)*(m+1)*(n+1));
	if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
    
    double* xB_d;
    cuda_ret = hipMalloc((void**) &xB_d, sizeof(double)*m);
	if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    double* objLine_d;
    cuda_ret = hipMalloc((void**) &objLine_d, sizeof(double)*n);
	if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    int* k_d;
    cuda_ret = hipMalloc((void**) &k_d, sizeof(int));
	if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    int* r_d;
    cuda_ret = hipMalloc((void**) &r_d, sizeof(int));
	if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    double* columnk_d;
    cuda_ret = hipMalloc((void**) &columnk_d, sizeof(double)*(m+1));
	if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    double* theta_d;
    cuda_ret = hipMalloc((void**) &columnk_d, sizeof(double)*(m+1));
	if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    // Copy tableau to Device
    cuda_ret = hipMemcpy(s_d, s_h, sizeof(double)*(m+1)*(n+1), hipMemcpyHostToDevice);
	if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device");

    cuda_ret = hipMemcpy(xB_d, xB_h, sizeof(double)*m, hipMemcpyHostToDevice);
	if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device");

    // Iterate until optimal is found

        // Copy first line of the Simplex tableau to host
        for (unsigned int i=0; i < m; i++) {objLine_d[i]= s_d[0][i+1] ;}

        cuda_ret = hipMemcpy(objLine_h, objLine_d, sizeof(double)*n, hipMemcpyDeviceToHost);
    	if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to host");
        
        hipDeviceSynchronize();

        // Find index of entering variable k
        double minValue = 1.0;
        int minIndex = 0;
        for (unsigned int i=0; i < m; i++) {
            if (objLine_h[i] < minValue) {
                minValue = objLine_h;
                minIndex = i;
            }
        }
        if (minValue>=0) {
            break;
        } else {
            k_h = minIndex + 1;
        }

        // Copy index k to device
        cuda_ret = hipMemcpy(k_d, k_h, sizeof(int), hipMemcpyHostToDevice);
	        if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device");

        hipDeviceSynchronize();

        // Kernel 1 to process ratio column
        const unsigned int THREADS_PER_BLOCK = 512;
        const unsigned int numBlocks1 = m/THREADS_PER_BLOCK + 1;
        dim3 gridDim(numBlocks1, 1, 1), blockDim(THREADS_PER_BLOCK, 1, 1);
        kernel1<<<gridDim, blockDim>>>(tab_d, theta_d, columnk_d, k_d);

        hipDeviceSynchronize();

        // Copy ratio column to host
        cuda_ret = hipMemcpy(theta_h, theta_d, sizeof(double)*(m+1), hipMemcpyDeviceToHost);
    	if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to host");
        
        // Find the index of the leaving variable r

        // Copy index r to device

        // Kernel 2 to update the line r of the Simplex tableau
        const unsigned int numBlocks2 = n/THREADS_PER_BLOCK + 1;
        dim3 gridDim(numBlocks2, 1, 1), blockDim(THREADS_PER_BLOCK, 1, 1);
        kernel2<<<gridDim, blockDim>>>(tab_d, columnk_d, k_d, r_d);

        hipDeviceSynchronize();

        // Kernel 3 to update Simplex tableau
        const unsigned int numBlocksX3 = m/THREADS_PER_BLOCK + 1;
        const unsigned int numBlocksY3 = n/THREADS_PER_BLOCK + 1;
        dim3 gridDim(numBlocksX3, numBlocksY3, 1), blockDim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
        kernel3<<<gridDim, blockDim>>>(tab_d, columnk_d, k_d, r_d);

        hipDeviceSynchronize();

        // Kernel 4 to Update column k of the Simplex Tableau
        const unsigned int numBlocks4 = n/THREADS_PER_BLOCK + 1;
        dim3 gridDim(numBlocks4, 1, 1), blockDim(THREADS_PER_BLOCK, 1, 1);
        kernel4<<<gridDim, blockDim>>>(tab_d, columnk_d, k_d, r_d);

        hipDeviceSynchronize();

    // Calculate optimal value and return it

}
