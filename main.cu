#include <stdio.h>
#include "kernel1.cu"
#include "kernel2.cu"
#include "kernel3.cu"
#include "kernel4.cu"
#include "support.h"
int main(int argc, char**argv) {


    unsigned int m; // number of constraints
    unsigned int n; // number of variables (no slack vars)
    unsigned int b; // b = highest value of constraint value 


    m = atoi(argv[1]);
    n = atoi(argv[2]);
    b = atoi(argv[3]);
    hipError_t cuda_ret;
    // Initialize host variables
    double* xB_h = (double*) malloc( sizeof(double)*m ); // index's of the variables in the Basis.
    for (unsigned int i=0; i < m; i++) { xB_h[i] = n+i; }
    
    double* xN_h = (double*) malloc( sizeof(double)*n ); // index's of the variables not in the Basis.
    for (unsigned int i=0; i < n; i++) { xN_h[i] = i; }
    
    double* cB_h = (double*) malloc( sizeof(double)*m ); // C values for the basic variables
    for (unsigned int i=0; i < m; i++) { cB_h[i] = 0; }

    double* cN_h = (double*) malloc( sizeof(double)*n ); // C values for the non-basic variables
    for (unsigned int i=0; i < n; i++) { cN_h[i] = (rand()%10-5); }

    double* b_h = (double*) malloc( sizeof(double)*m ); // Right hand side values 
    for (unsigned int i=0; i < m; i++) { b_h[i] = rand()%b; }
    
    double* svec = (double*) malloc( sizeof(double)*(m+1));
    



    //double* B_h = (double*) malloc( sizeof(double)*m*m ); // Constraint coefficents of basic variables
    double* B_h[m];
      for (int i = 0; i < m; i++) {
        B_h[i] = (double*)malloc(m * sizeof(double));
      }  
    for (unsigned int i=0; i < m; i++) { 
        for (unsigned int j=0; j < m; j++) { 
            if (i==j) {
                B_h[i][j] = 1;
            } else {
                B_h[i][j] = 0;
            }
        }
    }


    double* cBB	= (double*) malloc( sizeof(double)*m);
    for	(int i = 0; i < m; i++) {
    	double z = 0;
    	for (int j = 0; j < m; j++) {
            z += cB_h[j] * B_h[j][i];
     	}
	cBB[i] = z;
    }
    double z = 0;
    for (int i = 0; i < m; i++) {
    	z += cBB[i] * b_h[i];
    }
    svec[0] = z;
    for (int i = 1; i < m+1; i++) {
    	double sum = 0;
	for (int j = 0; j < m; j++) {
	    sum += B_h[i-1][j] * b_h[j];
	}
	svec[i] = sum;
    }   


       
    //double* N_h = (double*) malloc( sizeof(double)*m*n ); // Constraint coefficents of non-basic variables
    double* N_h[m];
    for (int i = 0; i < m; i++) {
        N_h[i] = (double*)malloc(n * sizeof(double));
    }
    printf("hi");
    fflush(stdout);
    for (unsigned int i=0; i < m; i++) { 
        for (unsigned int j=0; j < n; j++) { 
            N_h[i][j] = (rand()%10-5);
        }
    }


    //double* tab_h = (double*) malloc( sizeof(double)*(m+1)*(n+1)); // Simplex tableau
    double* tab_h[m+1];
    for (int i = 0; i < (m+1); i++) {
    	tab_h[i] = (double*) malloc((n+1) * sizeof(double));
    }
    for (int i = 0; i < (m+1); i++) {
    	tab_h[i][0] = xB_h[i];
    }
 
 
    // Assign vals for 0,0
    // Assign vals for m,0
    // Assign vals for 0,n
    // Assign vals for m,n
    double* smat[m+1];
    for (int i = 0; i < m+1; i++) {
        smat[i] = (double*) malloc(n * sizeof(double));
    }
    //update first row of smat
    for (int i = 0; i < n; i++) {
        double val = 0;
        for (int j = 0; j < m; j++) {
            val += cBB[j] * N_h[j][i];
        }
        smat[0][i] = val - cN_h[i];
    }
    for (int i = 1; i < m+1; i++) {

        for (int j = 0; j < n; j++) {
	    double val = 0;
	    for (int k = 0; k < m; k++) {
	    	
		val += B_h[i-1][k] * N_h[k][j];
		
	    	
	    }
	    smat[i][j] = val;
        }
    }

    //update tab
    for (int i = 0; i < m+1; i++) {
    	tab_h[i][0] = svec[i];
    }    
    
    for (int i = 0; i < m+1; i++) {
    	for (int j = 1; j < n+1; j++) {
	    tab_h[i][j] = smat[i][j];
	}   
    }	
    double* objLine_h = (double*) malloc( sizeof(double)*(n+1)); // Objective line used to determine k
    
    int k_h;
    
    int r_h;

    double* theta_h = (double*) malloc( sizeof(double)*(m+1)); // Ratio of right-hand side to k row

    // Allocate device variables
    double* tab_d;
    cuda_ret = hipMalloc((void**) &tab_d, sizeof(double)*(m+1)*(n+1));
	if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
    

 
    double* xB_d;
    cuda_ret = hipMalloc((void**) &xB_d, sizeof(double)*m);
	if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
    
    //double* objLine_d;
    //cuda_ret = hipMalloc((void**) &objLine_d, sizeof(double)*n);
	if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
	
    int k_d;
    //cuda_ret = hipMalloc((void**) &k_d, sizeof(int));
//	if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    int r_d;
    cuda_ret = hipMalloc((void**) &r_d, sizeof(int));
	if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    double* columnk_d;
    cuda_ret = hipMalloc((void**) &columnk_d, sizeof(double)*(m+1));
	if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    double* theta_d;
    cuda_ret = hipMalloc((void**) &theta_d, sizeof(double)*(m+1));
	if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    
    // Copy tableau to Device
    
    cuda_ret = hipMemcpy(tab_d, tab_h, sizeof(double)*(m+1)*(n+1), hipMemcpyHostToDevice);
	if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device");

    cuda_ret = hipMemcpy(xB_d, xB_h, sizeof(double)*m, hipMemcpyHostToDevice);
	if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device");
	
    // Iterate until optimal or infeasible/unbounded is found
    bool continueVar = true;
    while (continueVar == true) {
        // Copy first line of the Simplex tableau to host
        for (unsigned int i=0; i < n+1; i++) {objLine_h[i]= tab_h[0][i];}

        //cuda_ret = hipMemcpy(objLine_h, objLine_d, sizeof(double)*n, hipMemcpyDeviceToHost);
    	if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to host");
        
        hipDeviceSynchronize();

        // Find index of entering variable k
        double minValue = 1.0;
        int minIndex = 0;
        for (unsigned int i=1; i < n+1; i++) {
            if (objLine_h[i] < minValue) {
                minValue = objLine_h[i];
                minIndex = i;
            }
        }
        if (minValue>=0) {
            printf("Optimal Value");
            continueVar = false;
            break;
        } else {
            k_h = minIndex;
        }

        // Copy index k to device
     //   cuda_ret = hipMemcpy(k_d, k_h, sizeof(int), hipMemcpyHostToDevice);
//	        if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device");

        hipDeviceSynchronize();

        // Kernel 1 to process ratio column
        const unsigned int THREADS_PER_BLOCK = 512;
        const unsigned int numBlocks1 = m/THREADS_PER_BLOCK + 1;
        dim3 gridDim(numBlocks1, 1, 1), blockDim(THREADS_PER_BLOCK, 1, 1);
        kernel1<<<gridDim, blockDim>>>(tab_d, theta_d, columnk_d, k_d);

        hipDeviceSynchronize();

        // Copy ratio column to host
        cuda_ret = hipMemcpy(theta_h, theta_d, sizeof(double)*(m+1), hipMemcpyDeviceToHost);
    	if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to host");
        
        // Find the index of the leaving variable r
        minValue = 10000.0;
        minIndex = 0;
        for (unsigned int i=1; i < m+1; i++) {
            if (theta_h[i] < minValue && theta_h[i] > 0) {
                minValue = theta_h[i];
                minIndex = i;
            }
        }
        if (minValue==10000.0) {
            printf("unbounded");
            continueVar = false;
            break;
        } else {
            r_h = minIndex;
        }
        // Copy index r to device
       // cuda_ret = hipMemcpy(r_d, r_h, sizeof(int), hipMemcpyHostToDevice);
//	    if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device");

        // Kernel 2 to update the line r of the Simplex tableau
        const unsigned int numBlocks2 = n/THREADS_PER_BLOCK + 1;
        dim3 gridDim2(numBlocks2, 1, 1);
	dim3 blockDim2(THREADS_PER_BLOCK, 1, 1);
        kernel2<<<gridDim2, blockDim2>>>(tab_d, columnk_d, k_d, r_d);

        hipDeviceSynchronize();

        // Kernel 3 to update Simplex tableau
        const unsigned int numBlocksX3 = m/THREADS_PER_BLOCK + 1;
        const unsigned int numBlocksY3 = n/THREADS_PER_BLOCK + 1;
        dim3 gridDim3(numBlocksX3, numBlocksY3, 1), blockDim3(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
        kernel3<<<gridDim3, blockDim3>>>(tab_d, columnk_d, k_d, r_d);

        hipDeviceSynchronize();

        // Kernel 4 to Update column k of the Simplex Tableau
        const unsigned int numBlocks4 = n/THREADS_PER_BLOCK + 1;
        dim3 gridDim4(numBlocks4, 1, 1), blockDim4(THREADS_PER_BLOCK, 1, 1);
        kernel4<<<gridDim4, blockDim4>>>(tab_d, columnk_d, k_d, r_d);

        hipDeviceSynchronize();
    }
    // Calculate optimal value and return it
    //cuda_ret = hipMemcpy(objLine_h, objLine_d, sizeof(double)*n, hipMemcpyDeviceToHost);
    //if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to host");

}
