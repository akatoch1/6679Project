#include <stdio.h>

int main(int argc, char**argv) {
    // m = number of constraints
    // n = number of variables (no slack vars)
    // b = highest value of constraint value 
    
    // Initialize host variables
    double* xB_h = (double*) malloc( sizeof(double)*m );
    for (unsigned int i=0; i < m; i++) { xB_h[i] = n+1+i; }

    double* xN_h = (double*) malloc( sizeof(double)*n );
    for (unsigned int i=0; i < n; i++) { xB_h[i] = i+1; }

    double* cB_h = (double*) malloc( sizeof(double)*m );
    for (unsigned int i=0; i < m; i++) { cB_h[i] = 0; }

    float* cN_h = (float*) malloc( sizeof(float)*n );
    for (unsigned int i=0; i < n; i++) { cN_h[i] = (rand()%10-5); }

    float* b_h = (float*) malloc( sizeof(float)*m );
    for (unsigned int i=0; i < m; i++) { b_h[i] = rand()%b; }

    float* B_h = (float*) malloc( sizeof(float)*m*m );
    for (unsigned int i=0; i < m; i++) { 
        for (unsigned int j=0; j < m; j++) { 
            if (i==j) {
                B_h[i][j] = 1
            } else {
                B_h[i][j] = 0
            }
        }
    }

    float* N_h = (float*) malloc( sizeof(float)*m*n );
    for (unsigned int i=0; i < m; i++) { 
        for (unsigned int j=0; j < n; j++) { 
            B_h[i][j] = (rand()%10-5);
        }
    }

    float* s_h = (float*) malloc( sizeof(float)*(m+1)*(n+1));
    // Assign vals for 0,0
    // Assign vals for m,0
    // Assign vals for 0,n
    // Assign vals for m,n

    // Allocate device variables
    float* s_d;
    cuda_ret = hipMalloc((void**) &s_d, sizeof(float)*(m+1)*(n+1));
	if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
    
    float* xB_d;
    cuda_ret = hipMalloc((void**) &xB_d, sizeof(float)*m);
	if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    // Copy tableau to Device
    cuda_ret = hipMemcpy(s_d, s_h, sizeof(float)*(m+1)*(n+1), hipMemcpyHostToDevice);
	if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device");

    cuda_ret = hipMemcpy(xB_d, xB_h, sizeof(float)*m, hipMemcpyHostToDevice);
	if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device");

    // Iterate until optimal is found

        // Copy first line of the Simplex tableau to host

        // Find index of entering variable k

        // Copy index k to device

        // Kernel 1 to process ratio column

        // Copy ratio column to host

        // Find the index of the leaving variable r

        // Copy index r to device

        // Kernel 2 to update the line r of the Simplex tableau

        // Kernel 3 to update Simplex tableau

        // Kernel 4 to Update column k of the Simplex Tableau

        // Check if should stop

    // Calculate optimal value and return it

}
