#include <stdio.h>
#include "kernel1.cu"
#include "kernel2.cu"
#include "kernel3.cu"
#include "kernel4.cu"

int main(int argc, char**argv) {
    unsigned int m; // number of constraints
    unsigned int n; // number of variables (no slack vars)
    unsigned int b; // b = highest value of constraint value 
   
    m = atoi(argv[0]);
    n = atoi(argv[1]);
    b = atoi(argv[2]);
    
    // Initialize host variables
    double* xB_h = (double*) malloc( sizeof(double)*m ); // index's of the variables in the Basis.
    for (unsigned int i=0; i < m; i++) { xB_h[i] = n+i; }

    double* xN_h = (double*) malloc( sizeof(double)*n ); // index's of the variables not in the Basis.
    for (unsigned int i=0; i < n; i++) { xB_h[i] = i; }

    double* cB_h = (double*) malloc( sizeof(double)*m ); // C values for the basic variables
    for (unsigned int i=0; i < m; i++) { cB_h[i] = 0; }

    double* cN_h = (double*) malloc( sizeof(double)*n ); // C values for the non-basic variables
    for (unsigned int i=0; i < n; i++) { cN_h[i] = (rand()%10-5); }

    double* b_h = (double*) malloc( sizeof(double)*m ); // Right hand side values 
    for (unsigned int i=0; i < m; i++) { b_h[i] = rand()%b; }

    double* B_h = (double*) malloc( sizeof(double)*m*m ); // Constraint coefficents of basic variables
    for (unsigned int i=0; i < m; i++) { 
        for (unsigned int j=0; j < m; j++) { 
            if (i==j) {
                B_h[i][j] = 1;
            } else {
                B_h[i][j] = 0;
            }
        }
    }

    double* N_h = (double*) malloc( sizeof(double)*m*n ); // Constraint coefficents of non-basic variables
    for (unsigned int i=0; i < m; i++) { 
        for (unsigned int j=0; j < n; j++) { 
            B_h[i][j] = (rand()%10-5);
        }
    }

    double* tab_h = (double*) malloc( sizeof(double)*(m+1)*(n+1)); // Simplex tableau
    // Assign vals for 0,0
    // Assign vals for m,0
    // Assign vals for 0,n
    // Assign vals for m,n

    double* objLine_h = (double*) malloc( sizeof(double)*n); // Objective line used to determine k

    int* k_h = malloc(sizeof(int));
    
    int* r_h = malloc(sizeof(int));

    double* theta_h = (double*) malloc( sizeof(double)*(m+1)); // Ratio of right-hand side to k row

    // Allocate device variables
    double* tab_d;
    cuda_ret = hipMalloc((void**) &tab_d, sizeof(double)*(m+1)*(n+1));
	if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
    
    double* xB_d;
    cuda_ret = hipMalloc((void**) &xB_d, sizeof(double)*m);
	if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    double* objLine_d;
    cuda_ret = hipMalloc((void**) &objLine_d, sizeof(double)*n);
	if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    int* k_d;
    cuda_ret = hipMalloc((void**) &k_d, sizeof(int));
	if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    int* r_d;
    cuda_ret = hipMalloc((void**) &r_d, sizeof(int));
	if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    double* columnk_d;
    cuda_ret = hipMalloc((void**) &columnk_d, sizeof(double)*(m+1));
	if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    double* theta_d;
    cuda_ret = hipMalloc((void**) &columnk_d, sizeof(double)*(m+1));
	if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    // Copy tableau to Device
    cuda_ret = hipMemcpy(s_d, s_h, sizeof(double)*(m+1)*(n+1), hipMemcpyHostToDevice);
	if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device");

    cuda_ret = hipMemcpy(xB_d, xB_h, sizeof(double)*m, hipMemcpyHostToDevice);
	if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device");

    // Iterate until optimal or infeasible/unbounded is found
    bool continueVar = True;
    while (contunueVar == True) {
        // Copy first line of the Simplex tableau to host
        for (unsigned int i=0; i < n+1; i++) {objLine_d[i]= s_d[0][i] ;}

        cuda_ret = hipMemcpy(objLine_h, objLine_d, sizeof(double)*n, hipMemcpyDeviceToHost);
    	if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to host");
        
        hipDeviceSynchronize();

        // Find index of entering variable k
        double minValue = 1.0;
        int minIndex = 0;
        for (unsigned int i=1; i < n+1; i++) {
            if (objLine_h[i] < minValue) {
                minValue = objLine_h[i];
                minIndex = i;
            }
        }
        if (minValue>=0) {
            printf("Optimal Value")
            continueVar = False;
            break;
        } else {
            k_h = minIndex;
        }

        // Copy index k to device
        cuda_ret = hipMemcpy(k_d, k_h, sizeof(int), hipMemcpyHostToDevice);
	        if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device");

        hipDeviceSynchronize();

        // Kernel 1 to process ratio column
        const unsigned int THREADS_PER_BLOCK = 512;
        const unsigned int numBlocks1 = m/THREADS_PER_BLOCK + 1;
        dim3 gridDim(numBlocks1, 1, 1), blockDim(THREADS_PER_BLOCK, 1, 1);
        kernel1<<<gridDim, blockDim>>>(tab_d, theta_d, columnk_d, k_d);

        hipDeviceSynchronize();

        // Copy ratio column to host
        cuda_ret = hipMemcpy(theta_h, theta_d, sizeof(double)*(m+1), hipMemcpyDeviceToHost);
    	if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to host");
        
        // Find the index of the leaving variable r
        minValue = 10000.0;
        minIndex = 0;
        for (unsigned int i=1; i < m+1; i++) {
            if (theta_h[i] < minValue && theta_h[i] > 0) {
                minValue = theta_h[i];
                minIndex = i;
            }
        }
        if (minValue==10000.0) {
            printf("unbounded");
            continueVar = False;
            break;
        } else {
            r_h = minIndex;
        }
        // Copy index r to device
        cuda_ret = hipMemcpy(r_d, r_h, sizeof(int), hipMemcpyHostToDevice);
	    if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device");

        // Kernel 2 to update the line r of the Simplex tableau
        const unsigned int numBlocks2 = n/THREADS_PER_BLOCK + 1;
        dim3 gridDim(numBlocks2, 1, 1), blockDim(THREADS_PER_BLOCK, 1, 1);
        kernel2<<<gridDim, blockDim>>>(tab_d, columnk_d, k_d, r_d);

        hipDeviceSynchronize();

        // Kernel 3 to update Simplex tableau
        const unsigned int numBlocksX3 = m/THREADS_PER_BLOCK + 1;
        const unsigned int numBlocksY3 = n/THREADS_PER_BLOCK + 1;
        dim3 gridDim(numBlocksX3, numBlocksY3, 1), blockDim(THREADS_PER_BLOCK, THREADS_PER_BLOCK, 1);
        kernel3<<<gridDim, blockDim>>>(tab_d, columnk_d, k_d, r_d);

        hipDeviceSynchronize();

        // Kernel 4 to Update column k of the Simplex Tableau
        const unsigned int numBlocks4 = n/THREADS_PER_BLOCK + 1;
        dim3 gridDim(numBlocks4, 1, 1), blockDim(THREADS_PER_BLOCK, 1, 1);
        kernel4<<<gridDim, blockDim>>>(tab_d, columnk_d, k_d, r_d);

        hipDeviceSynchronize();
    }
    // Calculate optimal value and return it
    cuda_ret = hipMemcpy(objLine_h, objLine_d, sizeof(double)*n, hipMemcpyDeviceToHost);
    if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to host");

}
