#include <stdio.h>

int main(int argc, char**argv) {
    // m = number of constraints
    // n = number of variables (no slack vars)
    // b = highest value of constraint value 
    
    // Initialize host variables
    double* xB_h = (double*) malloc( sizeof(double)*m );
    for (unsigned int i=0; i < m; i++) { xB_h[i] = n+1+i; }

    double* xN_h = (double*) malloc( sizeof(double)*n );
    for (unsigned int i=0; i < n; i++) { xB_h[i] = i+1; }

    double* cB_h = (double*) malloc( sizeof(double)*m );
    for (unsigned int i=0; i < m; i++) { cB_h[i] = 0; }

    double* cN_h = (double*) malloc( sizeof(double)*n );
    for (unsigned int i=0; i < n; i++) { cN_h[i] = (rand()%10-5); }

    double* b_h = (double*) malloc( sizeof(double)*m );
    for (unsigned int i=0; i < m; i++) { b_h[i] = rand()%b; }

    double* B_h = (double*) malloc( sizeof(double)*m*m );
    for (unsigned int i=0; i < m; i++) { 
        for (unsigned int j=0; j < m; j++) { 
            if (i==j) {
                B_h[i][j] = 1;
            } else {
                B_h[i][j] = 0;
            }
        }
    }

    double* N_h = (double*) malloc( sizeof(double)*m*n );
    for (unsigned int i=0; i < m; i++) { 
        for (unsigned int j=0; j < n; j++) { 
            B_h[i][j] = (rand()%10-5);
        }
    }

    double* s_h = (double*) malloc( sizeof(double)*(m+1)*(n+1));
    // Assign vals for 0,0
    // Assign vals for m,0
    // Assign vals for 0,n
    // Assign vals for m,n

    double* objLine_h = (double*) malloc( sizeof(double)*n);

    int* k_h = malloc(sizeof(int));
    
    int* r_h = malloc(sizeof(int));

    double* columnk_h = (double*) malloc( sizeof(double)*(m+1));

    // Allocate device variables
    double* s_d;
    cuda_ret = hipMalloc((void**) &s_d, sizeof(double)*(m+1)*(n+1));
	if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
    
    double* xB_d;
    cuda_ret = hipMalloc((void**) &xB_d, sizeof(double)*m);
	if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    double* objLine_d;
    cuda_ret = hipMalloc((void**) &objLine_d, sizeof(double)*n);
	if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    int* k_d;
    cuda_ret = hipMalloc((void**) &k_d, sizeof(int));
	if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    int* r_d;
    cuda_ret = hipMalloc((void**) &r_d, sizeof(int));
	if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    double* columnk_d;
    cuda_ret = hipMalloc((void**) &columnk_d, sizeof(double)*(m+1));
	if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

    // Copy tableau to Device
    cuda_ret = hipMemcpy(s_d, s_h, sizeof(double)*(m+1)*(n+1), hipMemcpyHostToDevice);
	if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device");

    cuda_ret = hipMemcpy(xB_d, xB_h, sizeof(double)*m, hipMemcpyHostToDevice);
	if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device");

    // Iterate until optimal is found

        // Copy first line of the Simplex tableau to host
        for (unsigned int i=0; i < m; i++) {objLine_d[i]= s_d[0][i+1] ;}

        cuda_ret = hipMemcpy(objLine_h, objLine_d, sizeof(double)*n, hipMemcpyDeviceToHost);
    	if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to host");
        
        hipDeviceSynchronize();

        // Find index of entering variable k
        double minValue = 1.0;
        int minIndex = 0;
        for (unsigned int i=0; i < m; i++) {
            if (objLine_h[i] < minValue) {
                minValue = objLine_h;
                minIndex = i;
            }
        }
        if (minValue>=0) {
            break;
        } else {
            k_h = minIndex + 1;
        }

        // Copy index k to device
        cuda_ret = hipMemcpy(k_d, k_h, sizeof(int), hipMemcpyHostToDevice);
	        if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to device");

        hipDeviceSynchronize();

        // Kernel 1 to process ratio column

        // Copy ratio column to host
        cuda_ret = hipMemcpy(columnk_h, columnk_d, sizeof(double)*(m+1), hipMemcpyDeviceToHost);
    	if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to host");
        
        // Find the index of the leaving variable r

        // Copy index r to device

        // Kernel 2 to update the line r of the Simplex tableau

        // Kernel 3 to update Simplex tableau

        // Kernel 4 to Update column k of the Simplex Tableau

        // Check if should stop

    // Calculate optimal value and return it

}
