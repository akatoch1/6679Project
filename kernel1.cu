
#include <hip/hip_runtime.h>
__global__ void vecAddKernel(double **tab, double *theta, double *colk, int k) 
{
int i = blockDim.x * blockIdx.x + threadIdx.x;
double w = tab[i][k];
colk[i] = w;
theta[i] = tab[i][1]/w;

}