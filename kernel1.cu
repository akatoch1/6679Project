
#include <hip/hip_runtime.h>
__global__ void kernel1(double *tab, double *theta, double *colk, int k, int n) 
{
int i = blockDim.x * blockIdx.x + threadIdx.x;

double w = tab[i * (n+1) + k];

colk[i] = w;
theta[i] = tab[i * (n+1)]/w;

}