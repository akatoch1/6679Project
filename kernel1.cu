
#include <hip/hip_runtime.h>
__global__ void kernel1(double **tab, double *theta, double *colk, int k) 
{
int i = blockDim.x * blockIdx.x + threadIdx.x;
double w = tab[i][k];
colk[i] = w;
theta[i] = tab[i][0]/w;

}