
#include <hip/hip_runtime.h>
__global__ void kernel2(double **tab, double *colk, int k, int r)
{
int i = blockDim.x * blockIdx.x + threadIdx.x;
__shared__ double w;
if (threadIdx.x == 0) w = colk[r];
__syncthreads();
tab[r][i] = tab[r][i]/w;
}