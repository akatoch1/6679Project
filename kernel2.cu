
#include <hip/hip_runtime.h>
__global__ void kernel2(double *tab, double *colk, int k, int r, int n)
{
int i = blockDim.x * blockIdx.x + threadIdx.x;
__shared__ double w;
if (threadIdx.x == 0) w = colk[r];

__syncthreads();
//printf("%f ", w);

tab[r * (n+1) + i] = tab[r * (n+1) + i]/w;
}