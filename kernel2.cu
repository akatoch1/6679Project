
#include <hip/hip_runtime.h>
__global__ void kernel2(double *tab, double *colk, int k, int r, int n)
{

int i = blockDim.x * blockIdx.x + threadIdx.x;
if (i >= n+1) {
   return;
}
__shared__ double w;
if (threadIdx.x == 0) w = colk[r];

__syncthreads();


tab[r * (n+1) + i] = tab[r * (n+1) + i]/w;
}