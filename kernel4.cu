
#include <hip/hip_runtime.h>
__global__ void kernel4(double **tab, double *colk, int k, int r) 
{
int j = blockDim.x * blockIdx.x + threadIdx.x;
__shared__ double w;
if (threadIdx.x == 0) w = colk[r];
__syncthreads();
tab[j][k] = -colk[j]/w;

}