
#include <hip/hip_runtime.h>
__global__ void kernel4(double *tab, double *colk, int k, int r, int n) 
{
int j = blockDim.x * blockIdx.x + threadIdx.x;
__shared__ double w;
if (threadIdx.x == 0) w = colk[r];
__syncthreads();
tab[j * (n+1) + k] = -colk[j]/w;

}