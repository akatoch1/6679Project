
#include <hip/hip_runtime.h>
__global__ void kernel4(double *tab, double *colk, int k, int r, int n) 
{

int j = blockDim.x * blockIdx.x + threadIdx.x;

__shared__ double w;
if (threadIdx.x == 0) w = colk[r];
__syncthreads();
if (j == r) {
   tab[j*(n+1) + k] = 1/w;
}
else {

     tab[j * (n+1) + k] = -colk[j]/w;

}
}